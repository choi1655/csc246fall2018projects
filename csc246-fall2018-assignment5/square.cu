// Elapsed Real Time for input-4.txt: real	0m29.789s

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Size of the square we're looking for.
#define SQUARE_WIDTH 6
#define SQUARE_HEIGHT 6

// Maximum width of a row.  Makes it easier to allocate the whole
// grid contiguously.
#define MAX_WIDTH 16384

// Type used for a row of the grid.  Makes it easier to declare the
// grid as a pointer.
typedef char Row[ MAX_WIDTH ];

// Kernel, run by each thread to count complete squares in parallel.
__global__ void countSquares( int rows, int cols, bool report, Row *grid, int *output) {
  // Unique index for this worker.
  int r0 = blockDim.x * blockIdx.x + threadIdx.x;

  // Make sure I actually have something to work on.
  if ( r0 + SQUARE_HEIGHT - 1 < rows ) {
    int total = 0;
    //TODO logic goes here
    int startIdxCols = 0; //this is actually column
    int endIdxCols = startIdxCols + 6;
    int startIdxRows = r0; //so this is actually row
    int endIdxRows = startIdxRows + 6;
    int colidx = 0;
    int rowidx = 0;
    char square[6][6];
    while (endIdxCols <= cols) {
      //fill in square 2d array
      for (int i = startIdxRows; i < endIdxRows; i++) {
        for (int j = startIdxCols; j < endIdxCols; j++) {
          square[rowidx][colidx] = grid[i][j];
          // printf("%c %d %d\n", grid[i][j], i, j);
          // printf("%c\n", square[rowidx][colidx]);
          colidx++;
        }
        rowidx++;
        colidx = 0;
      }
      rowidx = 0;

      //at this point square is made
      //so check if it's valid square
      bool isValid = false;
      char *knownLetters = (char *)malloc(26 * sizeof(char));
      for (int i = 0; i < 26; i++) {
        *(knownLetters + i) = '*';
      }
      int counter = 0;
      for (int i = 0; i < 6; i++) {
        for (int j = 0; j < 6; j++) {
          bool exists = false;
          for (int k = 0; k <= counter; k++) {
            if (square[i][j] == *(knownLetters + k)) { //if letter is found
              exists = true;
            }
          }
          if (!exists) {
            *(knownLetters + counter) = square[i][j];
            counter++;
          }
        }
      }
      free(knownLetters);
      if (counter == 26) {
        isValid = true;
      }
      //End of boolean function

      if (isValid) {
        total++;
        if (report) {
          printf("%d %d\n", startIdxRows, startIdxCols);
        }
      }
      endIdxCols++;
      startIdxCols++;
    } //end of while loop

    *(output + r0) = total; //save the total number of squares to the unique index
    // printf("Total squares is %d with thread ID %d\n", total, r0);
  } //end of if statement
}

// Size of the grid of characters.
int rows, cols;

// Grid of letters.
Row *grid;

// Read the grid of characters.
void readGrid() {
  // Read grdi dimensions.
  scanf( "%d%d", &rows, &cols );
  if ( cols > MAX_WIDTH ) {
    fprintf( stderr, "Input grid is too wide.\n" );
    exit( EXIT_FAILURE );
  }

  // Make space to store the grid as a big, contiguous array.
  grid = (Row *) malloc( rows * sizeof( Row ) );

  // Read each row of the grid as a string, then copy everything
  // but the null terminator into the grid array.
  int rowCount = 0;
  char buffer[ MAX_WIDTH + 1 ];
  while ( rowCount < rows ) {
    scanf( "%s", buffer );
    memcpy( grid[ rowCount++ ], buffer, cols );
  }
}

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: square [report]\n" );
  exit( 1 );
}

int main( int argc, char *argv[] ) {
  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  // squareFound = false;
  readGrid();

  /** Array used to hold each squares found in each threads */
  // int *reportedSquares = (int *)malloc(rows * sizeof(int));

  // TODO Need to add code to allocate memory on the device and copy the grid
  // over.
  Row *rowGrid = NULL;
  hipMalloc((void **)&rowGrid, rows * sizeof(Row)); //allocate memory for 2 arrays
  hipMemcpy(rowGrid, grid, rows * sizeof(Row), hipMemcpyHostToDevice);

  // Block and grid dimensions.
  int threadsPerBlock = 250;
  // Round up.
  int blocksPerGrid = ( rows + threadsPerBlock - 1 ) / threadsPerBlock;

  // int *output = (int *)malloc(rows * sizeof(int));
  int *output = NULL;
  hipMalloc((void **)&output, rows * sizeof(int));
  hipMemset(output, 0x00, rows * sizeof(int));


  // printf("\n\n");

  // Run our kernel on these block/grid dimensions
  countSquares<<<blocksPerGrid, threadsPerBlock>>>( rows, cols, report, rowGrid, output);
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // TODO Need to add code to copy the results list back to the host and
  // add them up.
  // int *squareNums = NULL;
  int *mySquareNums = (int *)malloc(rows * sizeof(int));
  // cudaMalloc((void **)&squareNums, rows * sizeof(int));
  hipMemcpy(mySquareNums, output, rows * sizeof(int), hipMemcpyDeviceToHost);

  // for (int i = 0; i < rows; i++) {
  //   *(output + i) = 0;
  //   mySquareNums[i] = 0;
  // }
  int total = 0;
  for (int i = 0; i < rows; i++) {
    total += mySquareNums[i];
  }
  printf( "Squares: %d\n", total );

  hipFree(rowGrid);
  hipFree(output);
  free(mySquareNums);
  // Free memory on the device and the host.
  free( grid );

  hipDeviceReset(); //reset the device

  return 0;
}
